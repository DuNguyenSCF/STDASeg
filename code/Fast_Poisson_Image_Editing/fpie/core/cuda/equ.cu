#include "hip/hip_runtime.h"
#include "solver.h"
#include "utils.h"

CudaEquSolver::CudaEquSolver(int block_size)
    : maskbuf(NULL),
      imgbuf(NULL),
      block_size(block_size),
      cA(NULL),
      cimgbuf(NULL),
      cB(NULL),
      cX(NULL),
      tmp(NULL),
      EquSolver() {
  print_cuda_info();
  hipMalloc(&cerr, 3 * sizeof(float));
}

CudaEquSolver::~CudaEquSolver() {
  if (maskbuf != NULL) {
    delete[] maskbuf, imgbuf;
  }
  if (tmp != NULL) {
    hipFree(cA);
    hipFree(cB);
    hipFree(cX);
    hipFree(cimgbuf);
    hipFree(tmp);
  }
  hipFree(cerr);
}

py::array_t<int> CudaEquSolver::partition(py::array_t<int> mask) {
  auto arr = mask.unchecked<2>();
  int n = arr.shape(0), m = arr.shape(1);
  if (maskbuf != NULL) {
    delete[] maskbuf;
  }
  maskbuf = new int[n * m];
  int cnt = 0;
  for (int i = 0; i < n; ++i) {
    for (int j = 0; j < m; ++j) {
      if (arr(i, j) > 0) {
        maskbuf[i * m + j] = ++cnt;
      } else {
        maskbuf[i * m + j] = 0;
      }
    }
  }
  return py::array({n, m}, maskbuf);
}

void CudaEquSolver::post_reset() {
  if (cA != NULL) {
    delete[] imgbuf;
    hipFree(cA);
    hipFree(cB);
    hipFree(cX);
    hipFree(cimgbuf);
    hipFree(tmp);
  }
  imgbuf = new unsigned char[N * 3];
  hipMalloc(&cA, N * 4 * sizeof(int));
  hipMalloc(&cB, N * 3 * sizeof(float));
  hipMalloc(&cX, N * 3 * sizeof(float));
  hipMalloc(&cimgbuf, N * 3 * sizeof(unsigned char));
  hipMalloc(&tmp, N * 3 * sizeof(float));
  hipMemcpy(cA, A, N * 4 * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(cB, B, N * 3 * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(cX, X, N * 3 * sizeof(float), hipMemcpyHostToDevice);
}

__global__ void iter_equ_kernel(int N0, int N1, int* A, float* B, float* X) {
  int i = blockIdx.x * blockDim.x + threadIdx.x + N0;
  if (i < N1) {
    int off3 = i * 3;
    int4 id = (*((int4*)(A + i * 4))) * 3;
    float3 x = *((float3*)(B + off3));
    if (id.x) {
      x += *((float3*)(X + id.x));
    }
    if (id.y) {
      x += *((float3*)(X + id.y));
    }
    if (id.z) {
      x += *((float3*)(X + id.z));
    }
    if (id.w) {
      x += *((float3*)(X + id.w));
    }
    *((float3*)(X + off3)) = x / 4.0;
  }
}

__global__ void iter_shared_equ_kernel(int N0, int N1, int* A, float* B,
                                       float* X) {
  __shared__ float sX[4096 * 3];  // max shared size
  int i = blockIdx.x * blockDim.x + threadIdx.x + N0;
  if (i < N1) {
    int i0 = blockIdx.x * blockDim.x + N0;
    int i1 = (1 + blockIdx.x) * blockDim.x + N0;
    if (i1 > N1) i1 = N1;
    int off0 = i0 * 3;
    int off1 = i1 * 3;
    int off3 = i * 3;

    // load X to shared mem
    // sX[0..(i1 - i0), :] = X[i0..i1, :]
    *((float3*)(sX + off3 - off0)) = *((float3*)(X + off3));
    __syncthreads();

    int4 id = (*((int4*)(A + i * 4))) * 3;
    float3 x = *((float3*)(B + off3));
    if (id.x) {
      if (off0 <= id.x && id.x < off1) {
        x += *((float3*)(sX + id.x - off0));
      } else {
        x += *((float3*)(X + id.x));
      }
    }
    if (id.y) {
      if (off0 <= id.y && id.y < off1) {
        x += *((float3*)(sX + id.y - off0));
      } else {
        x += *((float3*)(X + id.y));
      }
    }
    if (id.z) {
      if (off0 <= id.z && id.z < off1) {
        x += *((float3*)(sX + id.z - off0));
      } else {
        x += *((float3*)(X + id.z));
      }
    }
    if (id.w) {
      if (off0 <= id.w && id.w < off1) {
        x += *((float3*)(sX + id.w - off0));
      } else {
        x += *((float3*)(X + id.w));
      }
    }
    *((float3*)(X + off3)) = x / 4.0;
  }
}

__global__ void error_equ_kernel(int N0, int N1, int* A, float* B, float* X,
                                 float* tmp) {
  int i = blockIdx.x * blockDim.x + threadIdx.x + N0;
  if (i < N1) {
    int off3 = i * 3;
    int4 id = (*((int4*)(A + i * 4))) * 3;
    float3 t = (*((float3*)(B + off3))) + (*((float3*)(X + id.x))) +
               (*((float3*)(X + id.y))) + (*((float3*)(X + id.z))) +
               (*((float3*)(X + id.w))) - (*((float3*)(X + off3))) * 4.0;
    *((float3*)(tmp + off3)) = fabs(t);
  }
}

__global__ void error_sum_equ_kernel(int N, int block_size, float* tmp,
                                     float* err) {
  __shared__ float sum_err[4096 * 3];  // max shared size
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  float3 err3 = make_float3(0.0, 0.0, 0.0);
  for (int i = id; i < N; i += block_size) {
    err3 += *((float3*)(tmp + i * 3));
  }
  *((float3*)(sum_err + id * 3)) = err3;
  __syncthreads();
  if (id == 0) {
    err3 = make_float3(0.0, 0.0, 0.0);
    for (int i = 0; i < block_size; ++i) {
      err3 += *((float3*)(sum_err + i * 3));
    }
    *(float3*)(err) = err3;
  }
}

__global__ void copy_X_equ_kernel(int N0, int N1, float* X,
                                  unsigned char* buf) {
  int i = blockIdx.x * blockDim.x + threadIdx.x + N0;
  if (i < N1) {
    buf[i] = X[i] < 0 ? 0 : X[i] > 255 ? 255 : X[i];
  }
}

std::tuple<py::array_t<unsigned char>, py::array_t<float>> CudaEquSolver::step(
    int iteration) {
  hipMemset(cerr, 0, 3 * sizeof(float));
  int grid_size = (N - 1 + block_size - 1) / block_size;
  for (int i = 0; i < iteration; ++i) {
    iter_equ_kernel<<<grid_size, block_size>>>(1, N, cA, cB, cX);
    // doesn't occur any numeric issue ...
    // hipDeviceSynchronize();
  }
  hipDeviceSynchronize();
  grid_size = (N * 3 - 3 + block_size - 1) / block_size;
  copy_X_equ_kernel<<<grid_size, block_size>>>(3, 3 * N, cX, cimgbuf);
  grid_size = (N - 1 + block_size - 1) / block_size;
  error_equ_kernel<<<grid_size, block_size>>>(1, N, cA, cB, cX, tmp);
  hipDeviceSynchronize();
  error_sum_equ_kernel<<<1, block_size>>>(N, block_size, tmp, cerr);
  hipDeviceSynchronize();

  hipMemcpy(err, cerr, 3 * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(imgbuf, cimgbuf, 3 * N * sizeof(unsigned char),
             hipMemcpyDeviceToHost);
  return std::make_tuple(py::array({N, 3}, imgbuf), py::array(3, err));
}
